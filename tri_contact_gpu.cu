#include "hip/hip_runtime.h"
﻿#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>

#include "vec3f_gpu.h"
#include "BBox_gpu.h"
#include "triFace_gpu.h"
#include "Node_gpu.h"
#include "build_gpu.h"

using namespace std;

hipError_t gCudaStatus;

#define CUDA_CHECK_CALL(fun, err_msg, return_code)					\
	gCudaStatus = fun;												\
	if(gCudaStatus != hipSuccess){									\
		printf("error_code%d: %s", gCudaStatus, err_msg);			\
		return return_code;											\
	}

int readObj(string objPath, int& vNum, int& fNum, triFace* faces, vec3f& min, vec3f& max){
	ifstream infile(objPath);
	if(!infile.is_open()){
		return -1;
	}
	string line;
	vNum = 0;
	fNum = 0;
	while(getline(infile, line)){
		if(line[0] == 'v' && line[1] == ' '){
			++vNum; 
		}
		else if(line[0] == 'f' && line[1] == ' '){
			++fNum;
		}
	}
	vec3f* point = new vec3f[vNum];
	vec3f* face = new vec3f[fNum];
	faces->points1 = new vec3f[fNum];
	faces->points2 = new vec3f[fNum];
	faces->points3 = new vec3f[fNum];
	faces->gravityPoints = new vec3f[fNum];
	faces->box = new BBox[fNum];
	infile.close();
	infile.open(objPath);
	if(!infile.is_open()){
		return -1;
	}
	int vNumIndex = 0, fNUmIndex = 0;
	string str;
	double d[3];
	while(getline(infile, line)){
		if(line[0] == 'v' && line[1] == ' '){
			istringstream instr(line);
			instr >> str >> d[0] >> d[1] >> d[2];
			// cout << str << " " << d1 << " " << d2 << " " << d3 << endl;
			point[vNumIndex].set_value(d[0], d[1], d[2]);
			++vNumIndex; 
			if(d[0] > max.x){
				max.x = d[0];
			}
			if(d[0] < min.x){
				min.x = d[0];
			}
			if(d[1] > max.y){
				max.y = d[1];
			}
			if(d[1] < min.y){
				min.y = d[1];
			}
			if(d[2] > max.z){
				max.z = d[2];
			}
			if(d[2] < min.z){
				min.z = d[2];
			}
		}
		else if(line[0] == 'f' && line[1] == ' '){
			istringstream instr(line);
			instr >> str;
			for(int i = 0; i < 3; ++i){
				instr >> str;
				d[i] = atof(str.c_str());
			}
			face[fNUmIndex].set_value(d[0], d[1], d[2]);
			++fNUmIndex;
		}
	}
	infile.close();
	for(int i = 0; i < fNum; ++i){
		faces->points1[i] = point[static_cast<int>(face[i].x) - 1];
		faces->points2[i] = point[static_cast<int>(face[i].y) - 1];
		faces->points3[i] = point[static_cast<int>(face[i].z) - 1];
		faces->gravityPoints[i] = (faces->points1[i] + faces->points2[i] + faces->points3[i]) / 3;
		faces->box[i] = getBox(*faces, i);
	}
	delete[] point;
	delete[] face;
	return 0;
}

int compare(const void* a, const void* b){
	return ((sorted*)a)->sortedMortonCode - ((sorted*)b)->sortedMortonCode;
}

__global__ void calclulateMorton3D(vec3f* gravityPoints, int fNum, unsigned int* sortedMortonCode, int* sortedObjectID){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i < fNum){
		sortedMortonCode[i] = morton3D(gravityPoints[i].x, gravityPoints[i].y, gravityPoints[i].z);
		sortedObjectID[i] = i;
	}
}
 
void buildBVH(triFace* faces, triFace* facesGPU, const int& fNum, const vec3f& min, const vec3f& max, Node** root){
	vec3f extend = max - min;
	// sorted* sortedCodeAndID = new sorted[fNum];
	// sorted* sortedCodeAndIDGPU;
	// hipMalloc((void**)&(sortedCodeAndIDGPU), fNum * sizeof(sorted));
	unsigned int* sortedMortonCode;
    int* sortedObjectID;
	hipMalloc((void**)&(sortedMortonCode), fNum * sizeof(unsigned int));
	hipMalloc((void**)&(sortedObjectID), fNum * sizeof(int));
	calclulateMorton3D<<<(fNum + 1023) / 1024, 1024>>>(facesGPU->gravityPoints, fNum, sortedMortonCode, sortedObjectID);
	hipDeviceSynchronize();
	// hipMemcpy(sortedCodeAndID, sortedCodeAndIDGPU, fNum * sizeof(sorted), hipMemcpyDeviceToHost);
	thrust::device_ptr<unsigned int> dev_key_ptr(sortedMortonCode);
	thrust::device_ptr<int> dev_data_ptr(sortedObjectID);
	thrust::sort_by_key(dev_key_ptr, dev_key_ptr + fNum, dev_data_ptr);
	sortedMortonCode = thrust::raw_pointer_cast(dev_key_ptr);
	sortedObjectID = thrust::raw_pointer_cast(dev_data_ptr);
	// qsort(sortedCodeAndID, fNum, sizeof(sorted), compare);
	// hipMemcpy(sortedCodeAndIDGPU, sortedCodeAndID, fNum * sizeof(sorted), hipMemcpyHostToDevice);
	generateHierarchy(facesGPU->box, sortedMortonCode, sortedObjectID, fNum, root);
	// hipFree(sortedCodeAndIDGPU);
	hipFree(sortedMortonCode);
	hipFree(sortedObjectID);
}

__device__ void traverseBVHGPU(vec3f* points1, vec3f* points2, vec3f* points3, BBox* bbox, const int& queryNum, Node** root){
	const BBox& box = bbox[queryNum];
	Node* rootPtr = *root;
	if(box_contact(box, *(rootPtr->getNodeBox()))){
		if(rootPtr->isLeaf()){
			int index = rootPtr->getIndex();
			if(queryNum < index && tri_contact(points1[queryNum], points2[queryNum], points3[queryNum], 
						   points1[index], points2[index], points3[index])){
				printf("%d %d\n", queryNum, index);
			}
		}
		else{
			Node* leftChild = rootPtr->leftChild;
			Node* rightChild = rootPtr->rightChild;
			traverseBVHGPU(points1, points2, points3, bbox, queryNum, &leftChild);
			traverseBVHGPU(points1, points2, points3, bbox, queryNum, &rightChild);
		}
	}
}

__global__ void traverseBVH(vec3f* points1, vec3f* points2, vec3f* points3, BBox* box, Node** root, int fNum){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i < fNum){
		traverseBVHGPU(points1, points2, points3, box, i, root);
	}
}


__global__ void freeOnDevice(Node** root){
	delete (*root);
	*root = nullptr;
}

int main(){
	size_t limit = 65536;//4096;	//set cuda stack size to avoid stackoverflow
	hipDeviceSetLimit(hipLimitStackSize, limit);
	hipDeviceGetLimit(&limit, hipLimitStackSize);
	printf("hipLimitStackSize: %llu\n", limit);
	limit = 536870912;
	hipDeviceSetLimit(hipLimitMallocHeapSize, limit);
	hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
	printf("hipLimitMallocHeapSize: %llu\n", limit);
	int vNum, fNum;
	clock_t start, end;
	vec3f min(100, 100, 100);
	vec3f max(-100, -100, -100);
	triFace* faces = new triFace;
	Node** root;
	hipMalloc((void **)&root, sizeof(Node*));
	start = clock();
	int res = readObj("flag-2000-changed.obj", vNum, fNum, faces, min, max);
	if(res){
		cout << "read obj failed" << endl;
	}
	end = clock();
	printf("read time=%fms\n",(double)(end-start) * 1000 /CLK_TCK);
	triFace* facesGPU = new triFace;
	// CUDA_CHECK_CALL(hipMalloc((void**)&(facesGPU), sizeof(triFace)), "hipMalloc failed!\n", -2);
	CUDA_CHECK_CALL(hipMalloc((void**)&(facesGPU->points1), fNum * sizeof(vec3f)), "hipMalloc failed!\n", -2);
	CUDA_CHECK_CALL(hipMalloc((void**)&(facesGPU->points2), fNum * sizeof(vec3f)), "hipMalloc failed!\n", -2);
	CUDA_CHECK_CALL(hipMalloc((void**)&(facesGPU->points3), fNum * sizeof(vec3f)), "hipMalloc failed!\n", -2);
	CUDA_CHECK_CALL(hipMalloc((void**)&(facesGPU->gravityPoints), fNum * sizeof(vec3f)), "hipMalloc failed!\n", -2);
	CUDA_CHECK_CALL(hipMalloc((void**)&(facesGPU->box), fNum * sizeof(BBox)), "hipMalloc failed!\n", -2);
	CUDA_CHECK_CALL(hipMemcpy(facesGPU->points1, faces->points1, fNum * sizeof(vec3f), hipMemcpyHostToDevice), "hipMemcpy points1 failed!\n", -2);
	CUDA_CHECK_CALL(hipMemcpy(facesGPU->points2, faces->points2, fNum * sizeof(vec3f), hipMemcpyHostToDevice), "hipMemcpy points2 failed!\n", -2);
	CUDA_CHECK_CALL(hipMemcpy(facesGPU->points3, faces->points3, fNum * sizeof(vec3f), hipMemcpyHostToDevice), "hipMemcpy points3 failed!\n", -2);
	CUDA_CHECK_CALL(hipMemcpy(facesGPU->gravityPoints, faces->gravityPoints, fNum * sizeof(vec3f), hipMemcpyHostToDevice), "hipMemcpy gravityPoints failed!\n", -2);
	CUDA_CHECK_CALL(hipMemcpy(facesGPU->box, faces->box, fNum * sizeof(BBox), hipMemcpyHostToDevice), "hipMemcpy box failed!\n", -2);
	start = clock();
	buildBVH(faces, facesGPU, fNum, min, max, root);
	end = clock();
	printf("build time=%fms\n",(double)(end-start) * 1000 /CLK_TCK);
	start = clock();
	traverseBVH<<<(fNum + 511) / 512, 512>>>(facesGPU->points1, facesGPU->points2, facesGPU->points3, facesGPU->box, root, fNum);
	hipDeviceSynchronize();
	hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
	end = clock();
	printf("traverse time=%fms\n",(double)(end-start) * 1000 /CLK_TCK);
	hipFree(facesGPU->points1);
	hipFree(facesGPU->points2);
	hipFree(facesGPU->points3);
	hipFree(facesGPU->gravityPoints);
	hipFree(facesGPU->box);
	delete facesGPU;
	freeOnDevice<<<1, 1>>>(root);
	hipFree(root);
	/*vec3f point11, point12, point13, point21, point22, point23;
	int collisionNum = 0;
	for(int i = 120914; i < fNum; ++i){
		for(int j = i + 1; j < fNum; ++j){
			point11 = faces->points1[i];
			point12 = faces->points2[i];
			point13 = faces->points3[i];
			point21 = faces->points1[j];
			point22 = faces->points2[j];
			point23 = faces->points3[j];
			if(tri_contact(point11, point12, point13, point21, point22, point23)){
				if(!(point11.equal_abs(point21) || point11.equal_abs(point22) || point11.equal_abs(point23) ||\
				     point12.equal_abs(point21) || point12.equal_abs(point22) || point12.equal_abs(point23) ||\
					 point13.equal_abs(point21) || point13.equal_abs(point22) || point13.equal_abs(point23))){
					++collisionNum;
					cout << "#self contact found at (" << i << "," << j << ")" << endl;
					cout << point11 << endl;
					cout << point12 << endl;
					cout << point13 << endl;
					cout << point21 << endl;
					cout << point22 << endl;
					cout << point23 << endl;
				}	
			}
		}
	}*/
	delete[] faces->points1;
	delete[] faces->points2;
	delete[] faces->points3;
	delete[] faces->gravityPoints;
	delete[] faces->box;
	delete faces;
	return 0;
}