#include "hip/hip_runtime.h"
﻿#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>

#include "vec3f_gpu.h"
#include "BBox_gpu.h"
#include "triFace_gpu.h"
#include "Node_gpu.h"
#include "build_gpu.h"

using namespace std;

hipError_t gCudaStatus;

#define CUDA_CHECK_CALL(fun, err_msg, return_code)					\
	gCudaStatus = fun;												\
	if(gCudaStatus != hipSuccess){									\
		printf("error_code%d: %s", gCudaStatus, err_msg);			\
		return return_code;											\
	}

int readObj(string objPath, int& vNum, int& fNum, triFace* faces, vec3f& min, vec3f& max){
	ifstream infile(objPath);
	if(!infile.is_open()){
		return -1;
	}
	string line;
	vNum = 0;
	fNum = 0;
	while(getline(infile, line)){
		if(line[0] == 'v' && line[1] == ' '){
			++vNum; 
		}
		else if(line[0] == 'f' && line[1] == ' '){
			++fNum;
		}
	}
	vec3f* point = new vec3f[vNum];
	vec3f* face = new vec3f[fNum];
	faces->points1 = new vec3f[fNum];
	faces->points2 = new vec3f[fNum];
	faces->points3 = new vec3f[fNum];
	faces->gravityPoints = new vec3f[fNum];
	faces->box = new BBox[fNum];
	infile.close();
	infile.open(objPath);
	if(!infile.is_open()){
		return -1;
	}
	int vNumIndex = 0, fNUmIndex = 0;
	string str;
	double d[3];
	while(getline(infile, line)){
		if(line[0] == 'v' && line[1] == ' '){
			istringstream instr(line);
			instr >> str >> d[0] >> d[1] >> d[2];
			// cout << str << " " << d1 << " " << d2 << " " << d3 << endl;
			point[vNumIndex].set_value(d[0], d[1], d[2]);
			++vNumIndex; 
			if(d[0] > max.x){
				max.x = d[0];
			}
			if(d[0] < min.x){
				min.x = d[0];
			}
			if(d[1] > max.y){
				max.y = d[1];
			}
			if(d[1] < min.y){
				min.y = d[1];
			}
			if(d[2] > max.z){
				max.z = d[2];
			}
			if(d[2] < min.z){
				min.z = d[2];
			}
		}
		else if(line[0] == 'f' && line[1] == ' '){
			istringstream instr(line);
			instr >> str;
			for(int i = 0; i < 3; ++i){
				instr >> str;
				d[i] = atof(str.c_str());
			}
			face[fNUmIndex].set_value(d[0], d[1], d[2]);
			++fNUmIndex;
		}
	}
	infile.close();
	for(int i = 0; i < fNum; ++i){
		faces->points1[i] = point[static_cast<int>(face[i].x) - 1];
		faces->points2[i] = point[static_cast<int>(face[i].y) - 1];
		faces->points3[i] = point[static_cast<int>(face[i].z) - 1];
		faces->gravityPoints[i] = (faces->points1[i] + faces->points2[i] + faces->points3[i]) / 3;
		faces->box[i] = getBox(*faces, i);
	}
	delete[] point;
	delete[] face;
	return 0;
}

int compare(const void* a, const void* b){
	return ((sorted*)a)->sortedMortonCode - ((sorted*)b)->sortedMortonCode;
}

__global__ void calclulateMorton3D(vec3f* gravityPoints, int fNum, unsigned int* sortedMortonCode, int* sortedObjectID, const vec3f min, const vec3f extend){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i < fNum){
		sortedMortonCode[i] = morton3D((gravityPoints[i].x - min.x) / extend.x, (gravityPoints[i].y - min.y) / extend.y, (gravityPoints[i].z - min.z) / extend.z);
		sortedObjectID[i] = i;
	}
}
 
void buildBVH(triFace* faces, triFace* facesGPU, const int& fNum, const vec3f& min, const vec3f& max, Node** root){
	vec3f extend = max - min;
	// sorted* sortedCodeAndID = new sorted[fNum];
	// sorted* sortedCodeAndIDGPU;
	// hipMalloc((void**)&(sortedCodeAndIDGPU), fNum * sizeof(sorted));
	unsigned int* sortedMortonCode;
    int* sortedObjectID;
	hipMalloc((void**)&(sortedMortonCode), fNum * sizeof(unsigned int));
	hipMalloc((void**)&(sortedObjectID), fNum * sizeof(int));
	calclulateMorton3D<<<(fNum + 1023) / 1024, 1024>>>(facesGPU->gravityPoints, fNum, sortedMortonCode, sortedObjectID, min, extend);
	hipDeviceSynchronize();
	// hipMemcpy(sortedCodeAndID, sortedCodeAndIDGPU, fNum * sizeof(sorted), hipMemcpyDeviceToHost);
	thrust::device_ptr<unsigned int> dev_key_ptr(sortedMortonCode);
	thrust::device_ptr<int> dev_data_ptr(sortedObjectID);
	thrust::sort_by_key(dev_key_ptr, dev_key_ptr + fNum, dev_data_ptr);
	sortedMortonCode = thrust::raw_pointer_cast(dev_key_ptr);
	sortedObjectID = thrust::raw_pointer_cast(dev_data_ptr);
	// qsort(sortedCodeAndID, fNum, sizeof(sorted), compare);
	// hipMemcpy(sortedCodeAndIDGPU, sortedCodeAndID, fNum * sizeof(sorted), hipMemcpyHostToDevice);
	generateHierarchy(facesGPU->box, sortedMortonCode, sortedObjectID, fNum, root);
	// hipFree(sortedCodeAndIDGPU);
	hipFree(sortedMortonCode);
	hipFree(sortedObjectID);
}

__device__ void traverseBVHGPU(vec3f* points1, vec3f* points2, vec3f* points3, BBox* bbox, const int& queryNum, Node* rootPtr){
	const BBox& box = bbox[queryNum];
	if(box_contact(box, *(rootPtr->getNodeBox()))){
		if(rootPtr->isLeaf()){
			int index = rootPtr->getIndex();
			if(queryNum < index && tri_contact(points1[queryNum], points2[queryNum], points3[queryNum], 
						   points1[index], points2[index], points3[index])){
				// printf("%d %d\n", queryNum, index);
			}
		}
		else{
			Node* leftChild = rootPtr->leftChild;
			Node* rightChild = rootPtr->rightChild;
			traverseBVHGPU(points1, points2, points3, bbox, queryNum, leftChild);
			traverseBVHGPU(points1, points2, points3, bbox, queryNum, rightChild);
		}
	}
}

__device__ void traverseIterativeBVHGPU( vec3f* points1, vec3f* points2, vec3f* points3, BBox* bbox, const int& queryNum, Node* root)
{
    // Allocate traversal stack from thread-local memory,
    // and push NULL to indicate that there are no postponed nodes.
    Node* stack[80];
    Node** stackPtr = stack;
    *stackPtr++ = NULL; // push

    // Traverse nodes starting from the root.
    do
    {
        // Check each child node for overlap.
        Node* childL = root->leftChild;
        Node* childR = root->rightChild;
        bool overlapL = ( box_contact(bbox[queryNum], *(childL->getNodeBox())));
        bool overlapR = ( box_contact(bbox[queryNum], *(childR->getNodeBox())));

        // Query overlaps a leaf node => report collision.
        if (overlapL && childL->isLeaf()){
			int index = childL->getIndex();
			if(queryNum < index && tri_contact(points1[queryNum], points2[queryNum], points3[queryNum], 
						   points1[index], points2[index], points3[index])){
				printf("%d %d\n", queryNum, childL->getIndex());
			}
		}

        if (overlapR && childR->isLeaf()){
			int index = childR->getIndex();

			if(queryNum < index && tri_contact(points1[queryNum], points2[queryNum], points3[queryNum], 
						   points1[index], points2[index], points3[index])){
				printf("%d %d\n", queryNum, childR->getIndex());
			}
		}
        // Query overlaps an internal node => traverse.
        bool traverseL = (overlapL && !childL->isLeaf());
        bool traverseR = (overlapR && !childR->isLeaf());

        if (!traverseL && !traverseR)
            root = *--stackPtr; // pop
        else
        {
            root = (traverseL) ? childL : childR;
            if (traverseL && traverseR)
                *stackPtr++ = childR; // push
        }
    }
    while (root != NULL);
}

__global__ void traverseBVH(vec3f* points1, vec3f* points2, vec3f* points3, BBox* box, Node** root, int fNum){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	Node* rootPrt = *root;
	if(i < fNum){
		// traverseBVHGPU(points1, points2, points3, box, i, rootPrt);
		traverseIterativeBVHGPU(points1, points2, points3, box, i, rootPrt);
	}
}


__global__ void freeOnDevice(Node** root){
	delete (*root);
	*root = nullptr;
}

int main(){
	size_t limit = 65536;//4096;	//set cuda stack size to avoid stackoverflow
	hipDeviceSetLimit(hipLimitStackSize, limit);
	hipDeviceGetLimit(&limit, hipLimitStackSize);
	printf("hipLimitStackSize: %llu\n", limit);
	limit = 536870912;
	hipDeviceSetLimit(hipLimitMallocHeapSize, limit);
	hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
	printf("hipLimitMallocHeapSize: %llu\n", limit);
	int vNum, fNum;
	clock_t start, end;
	vec3f min(100, 100, 100);
	vec3f max(-100, -100, -100);
	triFace* faces = new triFace;
	Node** root;
	hipMalloc((void **)&root, sizeof(Node*));
	start = clock();
	int res = readObj("flag-2000-changed.obj", vNum, fNum, faces, min, max);
	if(res){
		cout << "read obj failed" << endl;
	}
	end = clock();
	printf("read time=%fms\n",(double)(end-start) * 1000 /CLK_TCK);
	triFace* facesGPU = new triFace;
	// CUDA_CHECK_CALL(hipMalloc((void**)&(facesGPU), sizeof(triFace)), "hipMalloc failed!\n", -2);
	CUDA_CHECK_CALL(hipMalloc((void**)&(facesGPU->points1), fNum * sizeof(vec3f)), "hipMalloc failed!\n", -2);
	CUDA_CHECK_CALL(hipMalloc((void**)&(facesGPU->points2), fNum * sizeof(vec3f)), "hipMalloc failed!\n", -2);
	CUDA_CHECK_CALL(hipMalloc((void**)&(facesGPU->points3), fNum * sizeof(vec3f)), "hipMalloc failed!\n", -2);
	CUDA_CHECK_CALL(hipMalloc((void**)&(facesGPU->gravityPoints), fNum * sizeof(vec3f)), "hipMalloc failed!\n", -2);
	CUDA_CHECK_CALL(hipMalloc((void**)&(facesGPU->box), fNum * sizeof(BBox)), "hipMalloc failed!\n", -2);
	CUDA_CHECK_CALL(hipMemcpy(facesGPU->points1, faces->points1, fNum * sizeof(vec3f), hipMemcpyHostToDevice), "hipMemcpy points1 failed!\n", -2);
	CUDA_CHECK_CALL(hipMemcpy(facesGPU->points2, faces->points2, fNum * sizeof(vec3f), hipMemcpyHostToDevice), "hipMemcpy points2 failed!\n", -2);
	CUDA_CHECK_CALL(hipMemcpy(facesGPU->points3, faces->points3, fNum * sizeof(vec3f), hipMemcpyHostToDevice), "hipMemcpy points3 failed!\n", -2);
	CUDA_CHECK_CALL(hipMemcpy(facesGPU->gravityPoints, faces->gravityPoints, fNum * sizeof(vec3f), hipMemcpyHostToDevice), "hipMemcpy gravityPoints failed!\n", -2);
	CUDA_CHECK_CALL(hipMemcpy(facesGPU->box, faces->box, fNum * sizeof(BBox), hipMemcpyHostToDevice), "hipMemcpy box failed!\n", -2);
	start = clock();
	buildBVH(faces, facesGPU, fNum, min, max, root);
	end = clock();
	printf("build time=%fms\n",(double)(end-start) * 1000 /CLK_TCK);
	start = clock();
	traverseBVH<<<(fNum + 511) / 512, 512>>>(facesGPU->points1, facesGPU->points2, facesGPU->points3, facesGPU->box, root, fNum);
	hipDeviceSynchronize();
	end = clock();
	printf("traverse time=%fms\n",(double)(end-start) * 1000 /CLK_TCK);
	hipFree(facesGPU->points1);
	hipFree(facesGPU->points2);
	hipFree(facesGPU->points3);
	hipFree(facesGPU->gravityPoints);
	hipFree(facesGPU->box);
	delete facesGPU;
	freeOnDevice<<<1, 1>>>(root);
	hipFree(root);
	delete[] faces->points1;
	delete[] faces->points2;
	delete[] faces->points3;
	delete[] faces->gravityPoints;
	delete[] faces->box;
	delete faces;
	return 0;
}